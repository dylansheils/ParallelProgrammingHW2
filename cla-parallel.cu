#include "hip/hip_runtime.h"
/*********************************************************************/
//
// 02/01/2022: Revised Version for 32M bit adder with 32 bit blocks
//
/*********************************************************************/

#include "main.h"

//Touch these defines
#define input_size 8388608 // hex digits 
#define block_size 32
#define verbose 0

//Do not touch these defines
#define digits (input_size+1)
#define bits (digits*4)
#define ngroups bits/block_size
#define nsections ngroups/block_size
#define nsupersections nsections/block_size
#define nsupersupersections nsupersections/block_size

//Global definitions of the various arrays used in steps for easy access
int* gi; int* pi; int* ci; // Size: bits
int* ggj; int* gpj; int* gcj; // Size: ngroups
int* sgk; int* spk; int* sck; // Size: nsections
int* ssgl; int* sspl; int* sscl; // Size: nsupersupersections
int* sssgm; int* ssspm; int* ssscm; // Size: nsupersupersections
int* sumi; // Size: bits

int sumrca[bits] = {0};

//Integer array of inputs in binary form
int* bin1=NULL;
int* bin2=NULL;

//Character array of inputs in hex form
char* hex1=NULL;
char* hex2=NULL;

void read_input()
{
    char* in1 = (char*)calloc(input_size + 1, sizeof(char));
    char* in2 = (char*)calloc(input_size + 1, sizeof(char));

    if (1 != scanf("%s", in1))
    {
        printf("Failed to read input 1\n");
        exit(-1);
    }
    if (1 != scanf("%s", in2))
    {
        printf("Failed to read input 2\n");
        exit(-1);
    }

    hex1 = grab_slice_char(in1, 0, input_size + 1);
    hex2 = grab_slice_char(in2, 0, input_size + 1);

    free(in1);
    free(in2);
}
void ripple_carry_adder() {
    int clast = 0, cnext = 0;
    for (int i = 0; i < bits; i++) {
        cnext = (bin1[i] & bin2[i]) | ((bin1[i] | bin2[i]) & clast);
        sumrca[i] = bin1[i] ^ bin2[i] ^ clast;
        clast = cnext;
    }
}
void check_cla_rca() {
    for (int i = 0; i < bits; i++) {
        if (sumrca[i] != sumi[i]) {
            printf("Check: Found sumrca[%d] = %d, not equal to sumi[%d] = %d - stopping check here!\n",
                i, sumrca[i], i, sumi[i]);
            printf("bin1[%d] = %d, bin2[%d]=%d, gi[%d]=%d, pi[%d]=%d, ci[%d]=%d, ci[%d]=%d\n",
                i, bin1[i], i, bin2[i], i, gi[i], i, pi[i], i, ci[i], i - 1, ci[i - 1]);
            return;
        }
    }
    printf("Check Complete: CLA and RCA are equal\n");
}
void allocations() {
  hipMallocManaged(&sumi, bits * sizeof(int));
  hipMallocManaged(&gi, bits*sizeof(int)); hipMallocManaged(&pi, bits*sizeof(int)); hipMallocManaged(&ci, bits*sizeof(int));
  hipMallocManaged(&ggj, ngroups*sizeof(int)); hipMallocManaged(&gpj, ngroups*sizeof(int)); hipMallocManaged(&gcj, ngroups*sizeof(int));
  hipMallocManaged(&sgk, nsections*sizeof(int)); hipMallocManaged(&spk, nsections*sizeof(int)); hipMallocManaged(&sck, nsections*sizeof(int));
  hipMallocManaged(&ssgl, nsupersections*sizeof(int)); hipMallocManaged(&sspl, nsupersections*sizeof(int)); hipMallocManaged(&sscl, nsupersections*sizeof(int));
  hipMallocManaged(&sssgm, nsupersupersections*sizeof(int)); hipMallocManaged(&ssspm, nsupersupersections*sizeof(int)); hipMallocManaged(&ssscm, nsupersupersections*sizeof(int));
}
void deallocations() {
  hipFree(gi); hipFree(pi); hipFree(ggj); hipFree(sumi);
  hipFree(ggj); hipFree(gpj); hipFree(gcj);
  hipFree(sgk); hipFree(spk); hipFree(sck);
  hipFree(ssgl); hipFree(sspl); hipFree(sscl);
  hipFree(sssgm); hipFree(ssspm); hipFree(ssscm);
}

__global__ void compute_gp(int* gi, int* pi, int* bin1, int* bin2, int n) {
    int id = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (id < n) {
        gi[id] = bin1[id] & bin2[id];
        pi[id] = bin1[id] | bin2[id];
    }
}
__global__ void compute_forward(int sectionSize, int* f, int* s, int* t, int* fr) {
  int id = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (id < sectionSize) {
      int lstart = id * block_size;
      int sum = 0;
      for (int i = 0; i < block_size; i++) {
          int mult = f[i + lstart];
          for (int ii = block_size - 1; ii > i; ii--) {
              mult &= s[ii + lstart];
          }
          sum |= mult;
      }
      t[id] = sum;
      int mult = s[lstart];
      for (int i = 1; i < block_size; i++) {
          mult &= s[i + lstart];
      }
      fr[id] = mult;
  }
}
__global__ void compute_carry(int sectionSize, int* f, int* s, int* t, int* fr, int A, int B) {
    int id = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (id < sectionSize) {
        if (sectionSize == A) {
            int a = 0;
            if (id == 0) { a = 0; }
            else { a = f[id - 1]; }
            f[id] = s[id] | (t[id] & a);
        } else {
            int a = 0;
            if (id % B == B - 1) { a = f[id / B]; }
            else if (id != 0) { a = s[id - 1]; }
            s[id] = t[id] | (fr[id] & a);
        }
    }
}
__global__ void compute_sum(int n, int* ci, int* sumi, int* bin1, int* bin2) {
    int id = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (id < n) {
        int clast = 0;
        if (id == 0) { clast = 0; }
        else { clast = ci[id - 1]; }
        sumi[id] = bin1[id] ^ bin2[id] ^ clast;
    }
}

void cla() {
    int CUDAThreads = bits;
    int CUDABlock = 1024;
    int CUDAGrid = (int)((CUDAThreads / CUDABlock) + ((CUDAThreads % CUDABlock) != 0));
    hipMallocManaged(&bin1, bits * sizeof(int)); 
    hipMallocManaged(&bin2, bits * sizeof(int));

    allocations();

    compute_gp<<<CUDAGrid, CUDABlock>>>(gi, pi, bin1, bin2, bits);

    CUDAThreads = ngroups; CUDAGrid = (int)((CUDAThreads / CUDABlock) + ((CUDAThreads % CUDABlock) != 0));
    compute_forward<<<CUDAGrid, CUDABlock>>>(ngroups, gi, pi, ggj, gpj);
    
    CUDAThreads = nsections; CUDAGrid = (int)((CUDAThreads / CUDABlock) + ((CUDAThreads % CUDABlock) != 0));
    compute_forward << <CUDAGrid, CUDABlock >> > (nsections, ggj, gpj, sgk, spk);
    
    CUDAThreads = nsupersections; CUDAGrid = (int)((CUDAThreads / CUDABlock) + ((CUDAThreads % CUDABlock) != 0));
    compute_forward << <CUDAGrid, CUDABlock >> > (nsupersections, sgk, spk, ssgl, sspl);
    
    CUDAThreads = nsupersupersections; CUDAGrid = (int)((CUDAThreads / CUDABlock) + ((CUDAThreads % CUDABlock) != 0));
    compute_forward << <CUDAGrid, CUDABlock >> > (nsupersupersections, ssgl, sspl, sssgm, ssspm);
    compute_carry << <CUDAGrid, CUDABlock >> > (nsupersupersections, ssscm, sssgm, ssspm, NULL, nsupersupersections, block_size);
    
    CUDAThreads = nsupersections; CUDAGrid = (int)((CUDAThreads / CUDABlock) + ((CUDAThreads % CUDABlock) != 0));
    compute_carry << <CUDAGrid, CUDABlock >> > (nsupersections, ssscm, sscl, ssgl, sspl, nsupersupersections, block_size);
    
    CUDAThreads = nsections; CUDAGrid = (int)((CUDAThreads / CUDABlock) + ((CUDAThreads % CUDABlock) != 0));
    compute_carry << <CUDAGrid, CUDABlock >> > (nsections, sscl, sck, sgk, spk, nsupersupersections, block_size);
    
    CUDAThreads = ngroups; CUDAGrid = (int)((CUDAThreads / CUDABlock) + ((CUDAThreads % CUDABlock) != 0));
    compute_carry << <CUDAGrid, CUDABlock >> > (ngroups, sck, gcj, ggj, gpj, nsupersupersections, block_size);
    
    CUDAThreads = bits; CUDAGrid = (int)((CUDAThreads / CUDABlock) + ((CUDAThreads % CUDABlock) != 0));
    compute_carry << <CUDAGrid, CUDABlock >> > (bits, gcj, ci, gi, pi, nsupersupersections, block_size);
    compute_sum<<<CUDAGrid, CUDABlock>>>(bits, ci, sumi, bin1, bin2);

    hipDeviceSynchronize();
}

int main(int argc, char *argv[]) {
  int randomGenerateFlag = 1;
  int deterministic_seed = (1<<30) - 1;
  char* hexa=NULL;
  char* hexb=NULL;
  char* hexSum=NULL;
  char* int2str_result=NULL;
  unsigned long long start_time=clock_now(); // dummy clock reads to init
  unsigned long long end_time=clock_now();   // dummy clock reads to init

  if( nsupersupersections != block_size )
    {
      printf("Misconfigured CLA - nsupersupersections (%d) not equal to block_size (%d) \n",
	     nsupersupersections, block_size );
      return(-1);
    }
  
  if (argc == 2) {
    if (strcmp(argv[1], "-r") == 0)
      randomGenerateFlag = 1;
  }

  if (randomGenerateFlag == 0)
    {
      read_input();
    }
  else
    {
      srand( deterministic_seed );
      hex1 = generate_random_hex(input_size);
      hex2 = generate_random_hex(input_size);
    }
  
  hexa = prepend_non_sig_zero(hex1);
  hexb = prepend_non_sig_zero(hex2);
  hexa[digits] = '\0'; //double checking
  hexb[digits] = '\0';
  
  bin1 = gen_formated_binary_from_hex(hexa);
  bin2 = gen_formated_binary_from_hex(hexb);

  start_time = clock_now();
  cla();
  end_time = clock_now();

  printf("CLA Completed in %llu cycles\n", (end_time - start_time));

  start_time = clock_now();
  ripple_carry_adder();
  end_time = clock_now();

  printf("RCA Completed in %llu cycles\n", (end_time - start_time));

  check_cla_rca();

  if( verbose==1 )
    {
      int2str_result = int_to_string(sumi,bits);
      hexSum = revbinary_to_hex( int2str_result,bits);
    }

  // free inputs fields allocated in read_input or gen random calls
  free(int2str_result);
  free(hex1);
  free(hex2);
  
  // free bin conversion of hex inputs
  hipFree(bin2);
  hipFree(bin1);

  if( verbose==1 )
    {
      printf("Hex Input\n");
      printf("a   ");
      print_chararrayln(hexa);
      printf("b   ");
      print_chararrayln(hexb);
    }
  
  if ( verbose==1 )
    {
      printf("Hex Return\n");
      printf("sum =  ");
    }
  
  // free memory from prepend call
  free(hexa);
  free(hexb);

  if( verbose==1 )
    printf("%s\n",hexSum);
  
  free(hexSum);

  deallocations();
  
  return 1;
}